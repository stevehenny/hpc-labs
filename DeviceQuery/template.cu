#include "htk.h"

//@@ The purpose of this code is to become familiar with the submission
//@@ process. Do not worry if you do not understand all the details of
//@@ the code.

int main(int argc, char **argv) {
  int deviceCount;

  htkArg_read(argc, argv);

  hipGetDeviceCount(&deviceCount);

  htkTime_start(GPU, "Getting GPU Data."); //@@ start a timer

  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, dev);

    if (dev == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        htkLog(TRACE, "No CUDA GPU has been detected");
        return -1;
      } else if (deviceCount == 1) {
        //@@ htkLog is a provided logging API (similar to Log4J).
        //@@ The logging function htkLog takes a level which is either
        //@@ OFF, FATAL, ERROR, WARN, INFO, DEBUG, or TRACE and a
        //@@ message to be printed.
        htkLog(TRACE, "There is 1 device supporting CUDA");
      } else {
        htkLog(TRACE, "There are ", deviceCount,
              " devices supporting CUDA");
      }
    }

    htkLog(TRACE, "Device ", dev, " name: ", deviceProp.name);
    htkLog(TRACE, " Computational Capabilities: ", deviceProp.major, ".",
          deviceProp.minor);
    htkLog(TRACE, " Maximum global memory size: ",
          deviceProp.totalGlobalMem);
    htkLog(TRACE, " Maximum constant memory size: ",
          deviceProp.totalConstMem);
    htkLog(TRACE, " Maximum shared memory size per block: ",
          deviceProp.sharedMemPerBlock);
    htkLog(TRACE, " Maximum block dimensions: ",
          deviceProp.maxThreadsDim[0], " x ", deviceProp.maxThreadsDim[1],
          " x ", deviceProp.maxThreadsDim[2]);
    htkLog(TRACE, " Maximum grid dimensions: ", deviceProp.maxGridSize[0],
          " x ", deviceProp.maxGridSize[1], " x ",
          deviceProp.maxGridSize[2]);
    htkLog(TRACE, " Warp size: ", deviceProp.warpSize);
  }

  htkTime_stop(GPU, "Getting GPU Data."); //@@ stop the timer

  return 0;
}
