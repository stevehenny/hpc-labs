#include "hip/hip_runtime.h"

#include "htk.h"

#define htkCheck(stmt)                                                \
  do {                                                                \
    hipError_t err = stmt;                                           \
    if (err != hipSuccess) {                                         \
      htkLog(ERROR, "Failed to run stmt ", #stmt);                    \
      htkLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err)); \
      exit(1);                                                        \
    }                                                                 \
  } while (0)

// Compute C = A * B
// sgemm stands for single precision general matrix-matrix multiply
__global__ void sgemm(float *A, float *B, float *C,
                      int numARows, int numAColumns,
                      int numBRows, int numBColumns,
                      int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  int Row = blockIdx.y*blockDim.y+threadIdx.y;
  int Col = blockIdx.x*blockDim.x+threadIdx.x;
  if((Row < numCRows) && (Col < numCColumns)){
    float Pvalue = 0;
    for (int i = 0; i < numAColumns; i++){
        Pvalue += A[i+Row*numAColumns] * B[i*numBColumns+Col];
    }
    C[Row*numCColumns+Col] = Pvalue;
  }
												 

}

int main(int argc, char **argv) {
  htkArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C
  int numCColumns; // number of columns in the matrix C

  args = htkArg_read(argc, argv);
  if (args.inputCount != 2) {htkLog(ERROR, "Missing input"); return 1;}

  htkTime_start(IO, "Importing data and creating memory on host");
  hostA = (float *)htkImport(htkArg_getInputFile(args, 0), &numARows, &numAColumns);
  hostB = (float *)htkImport(htkArg_getInputFile(args, 1), &numBRows, &numBColumns);
  numCRows    = numARows;
  numCColumns = numBColumns;
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  htkTime_stop(IO, "Importing data and creating memory on host");
  htkLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  htkLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  htkLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);

  htkTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int Asize = sizeof(float) * numAColumns * numARows;
  int Bsize = sizeof(float) * numBColumns * numBRows;
  int Csize = sizeof(float) * numCColumns * numCRows;
  hipMalloc((void **)&deviceA, Asize);
  hipMalloc((void **)&deviceB, Bsize);
  hipMalloc((void **)&deviceC, Csize);

  htkTime_stop(GPU, "Allocating GPU memory.");

  htkTime_start(Copy, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, Asize, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, Bsize, hipMemcpyHostToDevice);
  hipMemcpy(deviceC, hostC, Csize, hipMemcpyHostToDevice);

  htkTime_stop(Copy, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(numCColumns/16.0), ceil(numCRows / 16.0), 1);
  dim3 DimBlock(16, 16, 1);

  htkTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  sgemm<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows,
  numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  htkTime_stop(Compute, "Performing CUDA computation");

  htkTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, Csize, hipMemcpyDeviceToHost);

  htkTime_stop(Copy, "Copying output memory to the CPU");

  htkTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  htkTime_stop(GPU, "Freeing GPU Memory");

  htkSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
